#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "hip/hip_runtime.h"

// Kernel function to add the elements of two arrays
__global__
void add(int n, float* x, float* y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

// function to add the elements of two arrays
// parallel GPU  // MultiThread by CUDA
__global__
void add_block(int n, float* x, float* y)
{
    //Thread index of the current Thread
    int index = threadIdx.x;
    // blockDim number of threads in the block
    int stride = blockDim.x;
    
    printf("ThreadIdx" + threadIdx.x);
    std::cout << "ThreadIdx" << threadIdx.x << std::endl;
    printf("BlockIdx" + blockIdx.x);

    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}


// parallel GPU  // MultiThread by CUDA in grid-stride loop https://colab.research.google.com/corgiredirector?site=https%3A%2F%2Fdeveloper.nvidia.com%2Fblog%2Fcuda-pro-tip-write-flexible-kernels-grid-stride-loops%2F
__global__
void add_grid(int n, float* x, float* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}


int main(void)
{
    int N = 1 << 20; // 1M elements

    /*
        float* x = new float[N];
        float* y = new float[N];
    */
    // Allocate Unified Memory -- accessible from CPU or GPU
    float* x, * y;
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));


    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }


    // Run kernel on 1M elements on the GPU
    add <<<1, 1 >>> (N, x, y);

    // 2nd Run kernel on 1M elements on the GPU
    add_block <<<1, 256 >> > (N, x, y);
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // 3rd Run kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add_grid <<<numBlocks, blockSize >>> (N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    /*
    delete[] x;
    delete[] y;
    */
    hipFree(x);
    hipFree(y);

    return 0;
}